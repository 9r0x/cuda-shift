#include "utils.h"

__host__ void catch_error(hipError_t err)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "[!] CUDA error: %s\n", hipGetErrorString(err));
        exit(1);
    }
}